
#include <hip/hip_runtime.h>
// Tests CUDA compilation targeting sm_next

// CC1 options level check.
// Check that by default we only supply sm_next CPU info without explicitly 
// overriding SM/PTX versions, and letting LLVM pick the defaults.
// RUN: %clang -### -c --target=x86_64-linux-gnu --cuda-device-only  \
// RUN:    --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda %s \
// RUN:    --cuda-gpu-arch=sm_next  2>&1 \
// RUN:   | FileCheck -check-prefixes=ARGS-COMMON,ARGS-ARCH %s
//
// Same, with explicitly set sm and PTX versions.
// RUN: %clang -### -c --target=x86_64-linux-gnu --cuda-device-only \
// RUN:    --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda %s \
// RUN:    --cuda-gpu-arch=sm_next --cuda-next-sm=111 --cuda-next-ptx=222  2>&1 \
// RUN:   | FileCheck -check-prefixes=ARGS-COMMON,ARGS-OVERRIDE %s

// Preprocessor level checks.
// RUN: %clang -dD -E --target=x86_64-linux-gnu --cuda-device-only -nocudainc \
// RUN:    --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda %s \
// RUN:    --cuda-gpu-arch=sm_next  2>&1 \
// RUN:   | FileCheck -check-prefixes=PP-COMMON,PP-ARCH %s
//
// Same, with explicitly set sm and PTX versions.
// RUN: %clang -dD -E --target=x86_64-linux-gnu --cuda-device-only -nocudainc \
// RUN:    --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda %s \
// RUN:    --cuda-gpu-arch=sm_next --cuda-next-sm=111 --cuda-next-ptx=222  2>&1 \
// RUN:   | FileCheck -check-prefixes=PP-COMMON,PP-OVERRIDE %s

// PTX level checks. 
// RUN: %clang -S --target=x86_64-linux-gnu --cuda-device-only -nocudainc -nocudalib \
// RUN:    --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda -o - %s \
// RUN:    --cuda-gpu-arch=sm_next  2>&1 \
// RUN:   | FileCheck -check-prefixes=PTX-ARCH %s
//
// Same, with explicitly set sm and PTX versions.
// RUN: %clang -S --target=x86_64-linux-gnu --cuda-device-only -nocudainc -nocudalib \
// RUN:      --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda -o - %s \
// RUN:      --cuda-gpu-arch=sm_next --cuda-next-sm=111 --cuda-next-ptx=222  2>&1 \
// RUN:   | FileCheck -check-prefixes=PTX-OVERRIDE %s


// ARGS-COMMON: "-cc1" "-triple" "nvptx64-nvidia-cuda"
// We need to pass specific SM version to CC1, so that preprocessor can set __CUDA_ARCH__ macro
// ARGS-ARCH-SAME: "--cuda-next-sm=52" "-mllvm" "--nvptx-next-sm=52"
// .. but we do not explicitly set PTX version, and carry on with the default for the detected CUDA SDK.
// ARGS-ARCH-NOT: --cuda-next-ptx=
// ARGS-ARCH-NOT: --nvptx-next-ptx=
// When we override SM and PTX versions, we explicitly set them for both clang and LLVM.
// ARGS-OVERRIDE-SAME: "--cuda-next-sm=111" "-mllvm" "--nvptx-next-sm=111" "--cuda-next-ptx=222" "-mllvm" "--nvptx-next-ptx=222"
// ARGS-COMMON-SAME: "-target-cpu" "sm_next"
// ARGS-COMMON-SAME: "-target-feature" "+ptx71"
// ARGS-COMMON-NEXT: ptxas
// ARGS-ARCH-SAME: "--gpu-name" "sm_52"
// ARGS-OVERRIDE-SAME: "--gpu-name" "sm_111"
//
//
// PP-COMMON:  #define __NVPTX__ 1
// PP-ARCH: #define __CUDA_ARCH__ 520
// PP-OVERRIDE: #define __CUDA_ARCH__  1110
//
//
// PTX-ARCH:  .version 8.5
// PTX-ARCH:  .target sm_52
// PTX-OVERRIDE:  .version 22.2
// PTX-OVERRIDE:  .target sm_111

